#include "hip/hip_runtime.h"
// -*- c++ -*-

#define NATIVE_LITTLE_ENDIAN 1

#include <atomic>
#include <cinttypes>
#include <cstddef>
#include <ctime>
#include <fstream>
#include <iostream>
#include <mutex>
#include <getopt.h>
#include <thread>
#include <vector>

#include "kadena-mine.hpp"
#include "optional.hpp"

namespace kadena {
namespace crypto {
namespace mining {
namespace cuda {

#define BLAKE2_INLINE inline

enum blake2s_constant
{
  BLAKE2S_BLOCKBYTES = 64,
  BLAKE2S_OUTBYTES   = 32,
  BLAKE2S_KEYBYTES   = 32,
  BLAKE2S_SALTBYTES  = 8,
  BLAKE2S_PERSONALBYTES = 8
};

//------------------------------------------------------------------------------
// kadena code

__device__
inline void memcpy__(void* dest0, const void* src0, uint32_t len) {
  const unsigned char* src = (const unsigned char*) src0;
  unsigned char* dest = (unsigned char*) dest0;

  while (len--) {
    *dest++ = *src++;
  }
}

__device__
inline void
memset__(void* buffer0, int c, size_t len) {
  char* buffer = (char*) buffer0;
  const char* end = buffer + len;
  for (char* p = buffer; p != end; ++p) {
    *p = c;
  }
}

__device__
inline void
secure_zero_memory(void* buffer0, size_t len) {
  return memset__(buffer0, 0, len);
}

// divide state into buffers puts contexts, so we can fit contexts into __shared__
typedef struct blake2s_context_gpu__
{
  uint32_t h[8];
  uint32_t t[2];
  uint32_t f[2];
} blake2s_gpu_context;

typedef struct blake2s_buffer_gpu__
{
  uint8_t  buf[BLAKE2S_BLOCKBYTES];
  uint32_t   buflen;
  uint32_t   outlen;
  uint64_t  last_node;   // 8 bits in original code but we need to make it word
                         // sized for padding.
} blake2s_gpu_buffer;

//------------------------------------------------------------------------------


/*
   BLAKE2 reference source code package - reference C implementations

   Copyright 2012, Samuel Neves <sneves@dei.uc.pt>.  You may use this under the
   terms of the CC0, the OpenSSL Licence, or the Apache Public License 2.0, at
   your option.  The terms of these licenses can be found at:

   - CC0 1.0 Universal : http://creativecommons.org/publicdomain/zero/1.0
   - OpenSSL license   : https://www.openssl.org/source/license.html
   - Apache 2.0        : http://www.apache.org/licenses/LICENSE-2.0

   More information about the BLAKE2 hash function can be found at
   https://blake2.net.
*/
#ifndef BLAKE2_IMPL_H
#define BLAKE2_IMPL_H

#if !defined(__cplusplus) && (!defined(__STDC_VERSION__) || __STDC_VERSION__ < 199901L)
  #if   defined(_MSC_VER)
    #define BLAKE2_INLINE __inline
  #elif defined(__GNUC__)
    #define BLAKE2_INLINE __inline__
  #else
    #define BLAKE2_INLINE
  #endif
#else
  #define BLAKE2_INLINE inline
#endif

__device__
static BLAKE2_INLINE uint32_t load32_( const void *src )
{
#if defined(NATIVE_LITTLE_ENDIAN)
  uint32_t w;
  memcpy__(&w, src, sizeof w);
  return w;
#else
  const uint8_t *p = ( const uint8_t * )src;
  return (( uint32_t )( p[0] ) <<  0) |
         (( uint32_t )( p[1] ) <<  8) |
         (( uint32_t )( p[2] ) << 16) |
         (( uint32_t )( p[3] ) << 24) ;
#endif
}

__device__
static BLAKE2_INLINE void store16( void *dst, uint16_t w )
{
#if defined(NATIVE_LITTLE_ENDIAN)
  memcpy__(dst, &w, sizeof w);
#else
  uint8_t *p = ( uint8_t * )dst;
  *p++ = ( uint8_t )w; w >>= 8;
  *p++ = ( uint8_t )w;
#endif
}

__device__
static BLAKE2_INLINE void store32( void *dst, uint32_t w )
{
#if defined(NATIVE_LITTLE_ENDIAN)
  memcpy__(dst, &w, sizeof w);
#else
  uint8_t *p = ( uint8_t * )dst;
  p[0] = (uint8_t)(w >>  0);
  p[1] = (uint8_t)(w >>  8);
  p[2] = (uint8_t)(w >> 16);
  p[3] = (uint8_t)(w >> 24);
#endif
}

__device__
static BLAKE2_INLINE uint32_t rotr32( const uint32_t w, const unsigned c )
{
  return ( w >> c ) | ( w << ( 32 - c ) );
}

#endif

//------------------------------------------------------------------------------

/*
   BLAKE2 reference source code package - reference C implementations

   Copyright 2012, Samuel Neves <sneves@dei.uc.pt>.  You may use this under the
   terms of the CC0, the OpenSSL Licence, or the Apache Public License 2.0, at
   your option.  The terms of these licenses can be found at:

   - CC0 1.0 Universal : http://creativecommons.org/publicdomain/zero/1.0
   - OpenSSL license   : https://www.openssl.org/source/license.html
   - Apache 2.0        : http://www.apache.org/licenses/LICENSE-2.0

   More information about the BLAKE2 hash function can be found at
   https://blake2.net.
*/
#ifndef BLAKE2_H
#define BLAKE2_H

#if defined(_MSC_VER)
#define BLAKE2_PACKED(x) __pragma(pack(push, 1)) x __pragma(pack(pop))
#else
#define BLAKE2_PACKED(x) x __attribute__((packed))
#endif

enum blake2b_constant
{
  BLAKE2B_BLOCKBYTES = 128,
  BLAKE2B_OUTBYTES   = 64,
  BLAKE2B_KEYBYTES   = 64,
  BLAKE2B_SALTBYTES  = 16,
  BLAKE2B_PERSONALBYTES = 16
};

typedef struct blake2s_state__
{
  uint32_t h[8];
  uint32_t t[2];
  uint32_t f[2];
  uint8_t  buf[BLAKE2S_BLOCKBYTES];
  size_t   buflen;
  size_t   outlen;
  uint8_t  last_node;
} blake2s_state;

typedef struct blake2b_state__
{
  uint64_t h[8];
  uint64_t t[2];
  uint64_t f[2];
  uint8_t  buf[BLAKE2B_BLOCKBYTES];
  size_t   buflen;
  size_t   outlen;
  uint8_t  last_node;
} blake2b_state;

typedef struct blake2sp_state__
{
  blake2s_state S[8][1];
  blake2s_state R[1];
  uint8_t       buf[8 * BLAKE2S_BLOCKBYTES];
  size_t        buflen;
  size_t        outlen;
} blake2sp_state;

typedef struct blake2bp_state__
{
  blake2b_state S[4][1];
  blake2b_state R[1];
  uint8_t       buf[4 * BLAKE2B_BLOCKBYTES];
  size_t        buflen;
  size_t        outlen;
} blake2bp_state;


BLAKE2_PACKED(struct blake2s_param__
              {
                uint8_t  digest_length; /* 1 */
                uint8_t  key_length;    /* 2 */
                uint8_t  fanout;        /* 3 */
                uint8_t  depth;         /* 4 */
                uint32_t leaf_length;   /* 8 */
                uint32_t node_offset;  /* 12 */
                uint16_t xof_length;    /* 14 */
                uint8_t  node_depth;    /* 15 */
                uint8_t  inner_length;  /* 16 */
                /* uint8_t  reserved[0]; */
                uint8_t  salt[BLAKE2S_SALTBYTES]; /* 24 */
                uint8_t  personal[BLAKE2S_PERSONALBYTES];  /* 32 */
});

typedef struct blake2s_param__ blake2s_param;

BLAKE2_PACKED(struct blake2b_param__
              {
                uint8_t  digest_length; /* 1 */
                uint8_t  key_length;    /* 2 */
                uint8_t  fanout;        /* 3 */
                uint8_t  depth;         /* 4 */
                uint32_t leaf_length;   /* 8 */
                uint32_t node_offset;   /* 12 */
                uint32_t xof_length;    /* 16 */
                uint8_t  node_depth;    /* 17 */
                uint8_t  inner_length;  /* 18 */
                uint8_t  reserved[14];  /* 32 */
                uint8_t  salt[BLAKE2B_SALTBYTES]; /* 48 */
                uint8_t  personal[BLAKE2B_PERSONALBYTES];  /* 64 */
});

typedef struct blake2b_param__ blake2b_param;

typedef struct blake2xs_state__
{
  blake2s_state S[1];
  blake2s_param P[1];
} blake2xs_state;

typedef struct blake2xb_state__
{
  blake2b_state S[1];
  blake2b_param P[1];
} blake2xb_state;

/* Padded structs result in a compile-time error */
enum {
  BLAKE2_DUMMY_1 = 1/(sizeof(blake2s_param) == BLAKE2S_OUTBYTES),
  BLAKE2_DUMMY_2 = 1/(sizeof(blake2b_param) == BLAKE2B_OUTBYTES)
};

#endif


//------------------------------------------------------------------------------
/*
   BLAKE2 reference source code package - reference C implementations

   Copyright 2012, Samuel Neves <sneves@dei.uc.pt>.  You may use this under the
   terms of the CC0, the OpenSSL Licence, or the Apache Public License 2.0, at
   your option.  The terms of these licenses can be found at:

   - CC0 1.0 Universal : http://creativecommons.org/publicdomain/zero/1.0
   - OpenSSL license   : https://www.openssl.org/source/license.html
   - Apache 2.0        : http://www.apache.org/licenses/LICENSE-2.0

   More information about the BLAKE2 hash function can be found at
   https://blake2.net.
*/

static __constant__ const uint32_t blake2s_IV[8] =
{
  0x6A09E667UL, 0xBB67AE85UL, 0x3C6EF372UL, 0xA54FF53AUL,
  0x510E527FUL, 0x9B05688CUL, 0x1F83D9ABUL, 0x5BE0CD19UL
};

static __constant__ const uint8_t blake2s_sigma[10][16] =
{
  {  0,  1,  2,  3,  4,  5,  6,  7,  8,  9, 10, 11, 12, 13, 14, 15 } ,
  { 14, 10,  4,  8,  9, 15, 13,  6,  1, 12,  0,  2, 11,  7,  5,  3 } ,
  { 11,  8, 12,  0,  5,  2, 15, 13, 10, 14,  3,  6,  7,  1,  9,  4 } ,
  {  7,  9,  3,  1, 13, 12, 11, 14,  2,  6,  5, 10,  4,  0, 15,  8 } ,
  {  9,  0,  5,  7,  2,  4, 10, 15, 14,  1, 11, 12,  6,  8,  3, 13 } ,
  {  2, 12,  6, 10,  0, 11,  8,  3,  4, 13,  7,  5, 15, 14,  1,  9 } ,
  { 12,  5,  1, 15, 14, 13,  4, 10,  0,  7,  6,  3,  9,  2,  8, 11 } ,
  { 13, 11,  7, 14, 12,  1,  3,  9,  5,  0, 15,  4,  8,  6,  2, 10 } ,
  {  6, 15, 14,  9, 11,  3,  0,  8, 12,  2, 13,  7,  1,  4, 10,  5 } ,
  { 10,  2,  8,  4,  7,  6,  1,  5, 15, 11,  9, 14,  3, 12, 13 , 0 } ,
};

__device__
static void blake2s_set_lastnode( blake2s_gpu_context *S )
{
  S->f[1] = (uint32_t)-1;
}

/* Some helper functions, not necessarily useful */
__device__
static int blake2s_is_lastblock( const blake2s_gpu_context *S )
{
  return S->f[0] != 0;
}

__device__
static void blake2s_set_lastblock( blake2s_gpu_context* S,
                                   blake2s_gpu_buffer *B )
{
  if( B->last_node ) blake2s_set_lastnode( S );
  S->f[0] = (uint32_t)-1;
}

__device__
static void blake2s_increment_counter( blake2s_gpu_context *S, const uint32_t inc )
{
  S->t[0] += inc;
  S->t[1] += ( S->t[0] < inc );
}

__device__
static void blake2s_init0( blake2s_gpu_context *S, blake2s_gpu_buffer* B )
{
  secure_zero_memory(S, sizeof(blake2s_gpu_context));
  secure_zero_memory(B, sizeof(blake2s_gpu_buffer));
  for(size_t i = 0; i < 8; ++i ) S->h[i] = blake2s_IV[i];
}

/* init2 xors IV with input parameter block */
__device__
int blake2s_init_param( blake2s_gpu_context *S, blake2s_gpu_buffer* B, const blake2s_param *P )
{
  const unsigned char *p = ( const unsigned char * )( P );
  size_t i;

  blake2s_init0( S, B );

  /* IV XOR ParamBlock */
  for( i = 0; i < 8; ++i )
    S->h[i] ^= load32_( &p[i * 4] );

  B->outlen = P->digest_length;
  return 0;
}


/* Sequential blake2s initialization */
__device__
int blake2s_init( blake2s_gpu_context *S, blake2s_gpu_buffer* B, size_t outlen )
{
  blake2s_param P[1];

  /* Move interval verification here? */
  if ( ( !outlen ) || ( outlen > BLAKE2S_OUTBYTES ) ) return -1;

  P->digest_length = (uint8_t)outlen;
  P->key_length    = 0;
  P->fanout        = 1;
  P->depth         = 1;
  store32( &P->leaf_length, 0 );
  store32( &P->node_offset, 0 );
  store16( &P->xof_length, 0 );
  P->node_depth    = 0;
  P->inner_length  = 0;
  /* memset(P->reserved, 0, sizeof(P->reserved) ); */
  memset( P->salt,     0, sizeof( P->salt ) );
  memset( P->personal, 0, sizeof( P->personal ) );
  return blake2s_init_param( S, B, P );
}

__device__
int blake2s_update( blake2s_gpu_context *S, blake2s_gpu_buffer* B,
                    const void *pin, size_t inlen );

#define G(r,i,a,b,c,d)                      \
  do {                                      \
    a = a + b + m[blake2s_sigma[r][2*i+0]]; \
    d = rotr32(d ^ a, 16);                  \
    c = c + d;                              \
    b = rotr32(b ^ c, 12);                  \
    a = a + b + m[blake2s_sigma[r][2*i+1]]; \
    d = rotr32(d ^ a, 8);                   \
    c = c + d;                              \
    b = rotr32(b ^ c, 7);                   \
  } while(0)

#define ROUND(r)                    \
  do {                              \
    G(r,0,v[ 0],v[ 4],v[ 8],v[12]); \
    G(r,1,v[ 1],v[ 5],v[ 9],v[13]); \
    G(r,2,v[ 2],v[ 6],v[10],v[14]); \
    G(r,3,v[ 3],v[ 7],v[11],v[15]); \
    G(r,4,v[ 0],v[ 5],v[10],v[15]); \
    G(r,5,v[ 1],v[ 6],v[11],v[12]); \
    G(r,6,v[ 2],v[ 7],v[ 8],v[13]); \
    G(r,7,v[ 3],v[ 4],v[ 9],v[14]); \
  } while(0)

#define COMPRESS(S, in, cst)                            \
  do {                                                  \
    uint32_t m[16];                                     \
    uint32_t v[16];                                     \
    size_t i;                                           \
                                                        \
    for( i = 0; i < 16; ++i ) {                         \
      m[i] = load32 ## cst( in + i * sizeof( m[i] ) );  \
    }                                                   \
                                                        \
    for( i = 0; i < 8; ++i ) {                          \
      v[i] = S->h[i];                                   \
    }                                                   \
                                                        \
    v[ 8] = blake2s_IV[0];                              \
    v[ 9] = blake2s_IV[1];                              \
    v[10] = blake2s_IV[2];                              \
    v[11] = blake2s_IV[3];                              \
    v[12] = S->t[0] ^ blake2s_IV[4];                    \
    v[13] = S->t[1] ^ blake2s_IV[5];                    \
    v[14] = S->f[0] ^ blake2s_IV[6];                    \
    v[15] = S->f[1] ^ blake2s_IV[7];                    \
                                                        \
    ROUND( 0 );                                         \
    ROUND( 1 );                                         \
    ROUND( 2 );                                         \
    ROUND( 3 );                                         \
    ROUND( 4 );                                         \
    ROUND( 5 );                                         \
    ROUND( 6 );                                         \
    ROUND( 7 );                                         \
    ROUND( 8 );                                         \
    ROUND( 9 );                                         \
                                                        \
    for( i = 0; i < 8; ++i ) {                          \
      S->h[i] = S->h[i] ^ v[i] ^ v[i + 8];              \
    }                                                   \
  } while(0)
__device__
static void blake2s_compress_( blake2s_gpu_context *S, const uint8_t* in)
{
  COMPRESS(S, in, _);
}

#undef G
#undef ROUND
#undef COMPRESS

#define UPD(S, B, in, inlen, cst_)                                     \
  if( inlen > 0 )                                                      \
  {                                                                    \
    size_t left = B->buflen;                                           \
    size_t fill = BLAKE2S_BLOCKBYTES - left;                           \
    if( inlen > fill )                                                 \
    {                                                                  \
      B->buflen = 0;                                                   \
      memcpy_ ## cst_( B->buf + left, in, fill ); /* Fill buffer */    \
      blake2s_increment_counter( S, BLAKE2S_BLOCKBYTES );              \
      blake2s_compress_( S, B->buf ); /* Compress */                   \
      in += fill; inlen -= fill;                                       \
      while(inlen > BLAKE2S_BLOCKBYTES) {                              \
        blake2s_increment_counter(S, BLAKE2S_BLOCKBYTES);              \
        blake2s_compress ## cst_( S, in );                             \
        in += BLAKE2S_BLOCKBYTES;                                      \
        inlen -= BLAKE2S_BLOCKBYTES;                                   \
      }                                                                \
    }                                                                  \
    memcpy_ ## cst_( B->buf + B->buflen, in, inlen );                  \
    B->buflen += inlen;                                                \
  }

__device__
int blake2s_update( blake2s_gpu_context *S, blake2s_gpu_buffer* B,
                    const void *pin, size_t inlen )
{
  const unsigned char * in = (const unsigned char *)pin;
  UPD(S, B, in, inlen, _);
  return 0;
}

#undef UPD

__device__
int blake2s_final( blake2s_gpu_context *S, blake2s_gpu_buffer* B,
                   void *out, size_t outlen )
{
  uint8_t buffer[BLAKE2S_OUTBYTES] = {0};
  size_t i;

  if( out == NULL || outlen < B->outlen )
    return -1;

  if( blake2s_is_lastblock( S ) )
    return -1;

  blake2s_increment_counter( S, ( uint32_t )B->buflen );
  blake2s_set_lastblock( S, B );
  memset( B->buf + B->buflen, 0, BLAKE2S_BLOCKBYTES - B->buflen ); /* Padding */
  blake2s_compress_( S, B->buf );

  for( i = 0; i < 8; ++i ) /* Output full hash to temp buffer */
    store32( buffer + sizeof( S->h[i] ) * i, S->h[i] );

  memcpy__( out, buffer, outlen );
  secure_zero_memory(buffer, sizeof(buffer));
  return 0;
}

//------------------------------------------------------------------------------
// kadena code
// kernel to test correctness of hash algo

__device__
void blake2s_one(const uint8_t* in, const size_t n,
                 uint8_t* out,
                 blake2s_gpu_context* ctx,
                 blake2s_gpu_buffer* buffer) {
  (void) blake2s_init(ctx, buffer, BLAKE2S_OUTBYTES);
  blake2s_update(ctx, buffer, in, n);
  blake2s_final(ctx, buffer, out, BLAKE2S_OUTBYTES);
}

__device__ void blake2s_kernel(
    const uint8_t* inputs,
    size_t* input_offsets,
    uint8_t* result_hashes,

    // these buffers have blockDim.x entries
    blake2s_gpu_context* contexts,
    blake2s_gpu_buffer* buffers) {
  const size_t local_id = threadIdx.x;
  const size_t global_id = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t input_sz = input_offsets[global_id + 1] - input_offsets[global_id];

  blake2s_gpu_buffer* buffer = &buffers[local_id];
  blake2s_gpu_context* ctx = &contexts[local_id];
  uint8_t* hash_out = result_hashes + (BLAKE2S_OUTBYTES * global_id);
  const uint8_t* in = &inputs[input_offsets[global_id]];
  blake2s_one(in, input_sz, hash_out, ctx, buffer);
}

static constexpr size_t INPUT_MAX = 1UL << 14;
__constant__ uint8_t g_target_hash[BLAKE2S_OUTBYTES];
__constant__ uint8_t g_input[INPUT_MAX];

// numbers stored little-endian, so compare from the end
__device__
inline bool lex_compare(uint8_t* hash0) {
  uint8_t* h = hash0 + BLAKE2S_OUTBYTES;
  const uint8_t* t = g_target_hash + BLAKE2S_OUTBYTES;
  bool ok = true;
  while (h != hash0) {
    --h; --t;
    ok &= (*h <= *t);
    if (*h != *t) break;
  }
  return ok;
}


__device__
inline uint8_t BLAKE2s_one_mine(const size_t n,
                                const uint64_t nonce,
                                blake2s_gpu_context* ctx,
                                blake2s_gpu_buffer* buffer) {
  uint8_t hash_out[BLAKE2S_OUTBYTES];
  (void) blake2s_init(ctx, buffer, BLAKE2S_OUTBYTES);
  {
    union {
      uint64_t as_val;
      uint8_t as_chars[sizeof(uint64_t)];
    } nonce_buf;
    nonce_buf.as_val = nonce;
    blake2s_update(ctx, buffer, nonce_buf.as_chars, sizeof(uint64_t));
  }
  size_t n_rest = n >= sizeof(uint64_t) ? n - sizeof(uint64_t) : 0;
  if (n_rest > 0) {
    blake2s_update(ctx, buffer, g_input + 8, n_rest);
  }
  blake2s_final(ctx, buffer, hash_out, BLAKE2S_OUTBYTES);
  return lex_compare(hash_out);
}

__global__ void BLAKE2s_mine(
    /* input length in bytes */
    const size_t input_length,

    /* starting nonce for the block */
    const uint64_t nonce_start,

    /* two words: is_found + nonce */
    uint64_t* results_buf,
    int num_nonces) {
  extern __shared__ uint8_t shared_bufs[];
  __shared__ uint8_t found;

  blake2s_gpu_context* contexts = (blake2s_gpu_context*) shared_bufs;
  blake2s_gpu_buffer* buffers;
  {
    blake2s_gpu_context* end_contexts = contexts + blockDim.x;
    buffers = (blake2s_gpu_buffer*) end_contexts;
  }

  const size_t local_id = threadIdx.x;
  if (local_id == 0) {
    found = 0;
  }

  __syncthreads();

  blake2s_gpu_buffer* buffer = &buffers[local_id];
  blake2s_gpu_context* ctx = &contexts[local_id];
  for (uint64_t global_id = blockDim.x * blockIdx.x + local_id;
       global_id < num_nonces;
       global_id += gridDim.x * blockDim.x) {
    if (found) {
      return;
    } else {
      bool ok = BLAKE2s_one_mine(
        input_length, nonce_start + global_id, ctx, buffer);
      if (ok) {
        results_buf[0] = 1;
        results_buf[1] = nonce_start + global_id;
        found = 1;
        return;
      }
    }
  }
}

namespace {
struct options {
  enum class mode { IMMEDIATE, CLIENT, SERVER };
  mode mode = mode::IMMEDIATE;
  // N.B. we use abstract namespace unix domain sockets, which are a big
  // improvement over the fs-based ones
  std::string unix_socket_namespace = "chainweb-gpu-miner0";
  std::string target_hash;
  nonstd::optional<uint64_t> starting_nonce;
  int local_workgroup_size = 380;
  int global_workgroup_size = 200000000UL;
  uint64_t starting_device = 0;
  uint64_t num_devices = 0;

  void fill_defaults() {
    auto r = []() -> uint32_t {
               return static_cast<uint32_t>(mrand48());
             };
    if (!starting_nonce) {
      const uint64_t nonce_a = r();
      const uint64_t nonce_b = r();
      const uint64_t nonce = (nonce_a << 32) | nonce_b;
      starting_nonce.emplace(nonce);
    }
  }
};

void usage(char** argv) {
  std::cout << "Usage: " << argv[0] << " {target_hash}\n" << R"raw(
Mine a block using blake2s. The block bytes will be accepted on stdin, with
the first eight bytes of the block left unspecified (the "nonce"). The miner
will search for a nonce that will make the total block hash (evaluated as a
little-endian 256-bit integer) lower than the target_hash.

The target hash may also be passed via environment variable as TARGET_HASH.
Note that {target_hash} is ignored in --server mode: the target hash is passed
to a server over the socket.

Upon success, the miner will output the nonce to stdout as a hexified string.

Options:

  --help                 print this message
  --starting-nonce k     start nonce search at (decimal integer).
  --local-wg-size k      size of local work group (default 380)
  --global-wg-size k     size of global work group (default 200000000)
  --starting-device i    start at device i
  --num-devices n        operate on n devices
  --server               start in server mode (see --unix-domain-ns)
  --client               start in client mode
  --unix-domain-ns ns    use unix domain namespace 'ns' for client/server mode

)raw";
  exit(1);
}

options parse_options(int argc, char** argv) {
  options out;
  {
    const char* env = getenv("TARGET_HASH");
    if (env != nullptr) out.target_hash = env;
  }
  int c = 0;

  while (true) {
    int option_index = 0;
    static struct option long_options[] = {
      {"help",            no_argument,       0,  0 },
      {"starting-nonce",  required_argument, 0,  0 },
      {"local-wg-size",   required_argument, 0,  0 },
      {"global-wg-size",  required_argument, 0,  0 },
      {"starting-device", required_argument, 0,  0 },
      {"num-devices",     required_argument, 0,  0 },
      {"client",          no_argument,       0,  0 },
      {"server",          no_argument,       0,  0 },
      {"unix-domain-ns",  required_argument, 0,  0 },
      {0,                 0,                 0,  0 }
    };

    c = getopt_long(argc, argv, "", long_options, &option_index);
    if (c == -1) break;

    switch (c) {
    case 0:
      switch (option_index) {
      case 0: usage(argv); break;
      case 1:
        out.starting_nonce.emplace(parse_nonce(optarg));
        break;
      case 2:
        out.local_workgroup_size = atoi(optarg);
        break;
      case 3:
        out.global_workgroup_size = atoi(optarg);
        break;
      case 4:
        out.starting_device = atoi(optarg);
        break;
      case 5:
        out.num_devices = atoi(optarg);
        break;
      case 6:
        out.mode = options::mode::CLIENT;
        break;
      case 7:
        out.mode = options::mode::SERVER;
        break;
      case 8:
        out.unix_socket_namespace = optarg;
        break;
      default:  // impossible
        break;
      }
      break;

    default:
      break;
    }
  }

  if (optind == argc - 1) {
    out.target_hash = argv[optind];
  }

  out.fill_defaults();
  std::string s = hex_decode(out.target_hash);
  out.target_hash = std::move(s);
  return out;
}

static constexpr int NSTREAMS = 3;
struct mining_buffers {
  std::array<uint8_t*, NSTREAMS> results_bufs = {};
  std::array<uint8_t*, NSTREAMS> results_host_bufs = {};
  std::array<hipStream_t, NSTREAMS> stream;
  std::array<nonstd::optional<uint64_t>, NSTREAMS> active;
  int which = 0;

  mining_buffers() {
    for (int i = 0; i < NSTREAMS; ++i) {
      hipStreamCreate(&stream[i]);
    }
  }
  mining_buffers(const mining_buffers&) = delete;
  mining_buffers(mining_buffers&& o) {
    results_bufs = std::move(o.results_bufs);
    results_host_bufs = std::move(o.results_host_bufs);
    stream = std::move(o.stream);
    o.clear();
  }

  void flip() {
    int n = next();
    which = n;
  }

  int next() {
    return (which + 1) % NSTREAMS;
  }

  uint8_t* buf(int i) { return results_bufs[i]; }
  uint8_t* host_buf(int i) { return results_host_bufs[i]; }

  mining_buffers& operator=(const mining_buffers&) = delete;
  mining_buffers& operator=(mining_buffers&& o) = delete;

  ~mining_buffers() {
    free_and_clear();
  }

private:
  void free_and_clear() {
    for (auto* p : results_bufs) if (p) hipFree(p);
    for (auto* p : results_host_bufs) if (p) hipHostFree(p);
    for (auto s : stream) if (s) hipStreamDestroy(s);
    clear();
  }

  void clear() {
    for (auto& s : stream) s = nullptr;
    for (auto& p : results_bufs) p = nullptr;
    for (auto& p : results_host_bufs) p = nullptr;
  }
};

mining_buffers
alloc_buffers(const std::string& input,
              const std::string& target_hash) {
  mining_buffers out;
  if (input.size() > INPUT_MAX) {
    throw std::runtime_error("input too large");
  }
  hipMemcpyToSymbol(HIP_SYMBOL(g_target_hash), target_hash.data(), target_hash.size());
  hipMemcpyToSymbol(HIP_SYMBOL(g_input), input.data(), input.size());
  for (int i = 0; i < NSTREAMS; ++i) {
    hipHostMalloc(&out.results_host_bufs[i], 2 * sizeof(uint64_t));
    hipMalloc(&out.results_bufs[i], 2 * sizeof(uint64_t));
  }
  return out;
}

std::mutex g_mutex;

void
cpu_thread(const options& options,
           const std::string& target_hash,
           const std::string& blockbytes,
           mining_synchronization& sync,
           int device_number) {
  auto fail_with_msg =
    [&](const std::string& msg) {
      std::lock_guard<std::mutex> l(g_mutex);
      DBG() << "GPU " << device_number
            << ": got exception (exiting): "
            << msg
            << "\n";
      sync.terminate_cancelled();
    };
  try {
    {
      std::lock_guard<std::mutex> l(g_mutex);
      DBG() << "GPU "
            << device_number
            << ": starting mining\n";
    }
    hipSetDevice(device_number);

    mining_buffers buffers = alloc_buffers(
      blockbytes,
      target_hash);
    uint64_t group_size = options.local_workgroup_size;
    uint64_t num_per_thread = 256;

    const dim3 grid(
      (options.global_workgroup_size +
       (num_per_thread*group_size)-1)/(num_per_thread*group_size));
    const dim3 block(group_size);

    while (!sync.finished()) {
      buffers.flip();             // flip double buffer
      uint64_t nonce_start = sync.next_nonce();
      buffers.active[buffers.which] = nonce_start;
      hipMemsetAsync(buffers.buf(buffers.which),
                      0,
                      2 * sizeof(uint64_t),
                      buffers.stream[buffers.which]);
      auto cuda_err = hipGetLastError();
      if (cuda_err) {
        throw std::runtime_error(hipGetErrorString(cuda_err));
      }
      BLAKE2s_mine<<<
        grid, block,
          sizeof(blake2s_gpu_context)*group_size +
          sizeof(blake2s_gpu_buffer)*group_size,
          buffers.stream[buffers.which]
          >>>(
            blockbytes.size(),
            nonce_start,
            reinterpret_cast<uint64_t*>(buffers.buf(buffers.which)),
            options.global_workgroup_size);
      cuda_err = hipGetLastError();
      if (cuda_err) {
        throw std::runtime_error(hipGetErrorString(cuda_err));
      }
      hipMemcpyAsync(buffers.host_buf(buffers.which),
                      buffers.buf(buffers.which),
                      2 * sizeof(uint64_t),
                      hipMemcpyDeviceToHost,
                      buffers.stream[buffers.which]);
      int next = buffers.next();
      if (buffers.active[next]) {
        // wait on previously enqueued work
        hipStreamSynchronize(buffers.stream[next]);
        uint64_t* buf = reinterpret_cast<uint64_t*>(buffers.host_buf(next));
        uint64_t found = buf[0];
        if (found) {
          std::lock_guard<std::mutex> l(g_mutex);
          DBG() << "GPU " << device_number
                << ": found winning nonce "
                << std::hex
                << buf[1]
                << "\n";
          // sync device before we tear down state.
          hipDeviceSynchronize();
          sync.terminate_success(buf[1]);
          return;
        } else {
          std::lock_guard<std::mutex> l(g_mutex);
          DBG() << "GPU " << device_number
                << ": no matching nonces starting at "
                << nonce_to_string(*buffers.active[next])
                << "\n";
        }
        buffers.active[next] = {};
      }
    }
  } catch (const std::exception& e) {
    fail_with_msg(e.what());
  } catch (...) {
    fail_with_msg("unknown exception");
  }
}
}   // namespace
}   // namespace cuda
}   // namespace mining
}   // namespace crypto
}   // namespace kadena


int
main(int argc, char** argv) {
  try {
    using namespace kadena::crypto;
    using namespace kadena::crypto::mining;
    using namespace kadena::crypto::mining::cuda;
    srand48_seeder seed;
    options options = parse_options(argc, argv);
    if (options.mode == options::mode::SERVER ||
        options.mode == options::mode::IMMEDIATE) {
      set_debug_timestamps(false);
      int num_devices = 0;
      (void) hipGetDeviceCount(&num_devices);
      DBG() << "Cuda reports " << num_devices << " devices.\n";
      if (options.num_devices != 0) {
        num_devices = std::min(
          num_devices - options.starting_device,
          options.num_devices);
      }

      if (num_devices == 0) {
        throw std::runtime_error("No devices found.");
      }
      options.num_devices = num_devices;

      DBG() << "Running " << num_devices << " devices, starting at "
            << options.starting_device << "\n";
    }

    if (options.mode == options::mode::IMMEDIATE) {
      run_immediate_mode(argc, argv, options, &usage, &cpu_thread);
    } else if (options.mode == options::mode::SERVER) {
      run_server_mode(options, &cpu_thread);
    } else {
      run_client_mode(argc, argv, options, &usage, &cpu_thread);
    }
  } catch (const std::exception& err) {
    std::cerr << "Caught exception at top level: "
              << err.what()
              << "\n";
    exit(1);
  }
  return 0;
}
